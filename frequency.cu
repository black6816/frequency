#include "hip/hip_runtime.h"
/*
Detected 1 CUDA Capable device(s)

Device 0: "GeForce GT 320M"
  CUDA Driver Version / Runtime Version          5.0 / 5.0
  CUDA Capability Major/Minor version number:    1.2
  Total amount of global memory:                 1024 MBytes (1073741824 bytes)
  ( 3) Multiprocessors x (  8) CUDA Cores/MP:    24 CUDA Cores
  GPU Clock rate:                                1100 MHz (1.10 GHz)
  Memory Clock rate:                             790 Mhz
  Memory Bus Width:                              128-bit
  Max Texture Dimension Size (x,y,z)             1D=(8192), 2D=(65536,32768), 3D=(2048,2048,2048)
  Max Layered Texture Size (dim) x layers        1D=(8192) x 512, 2D=(8192,8192) x 512
  Total amount of constant memory:               65536 bytes
  Total amount of shared memory per block:       16384 bytes
  Total number of registers available per block: 16384
  Warp size:                                     32
  Maximum number of threads per multiprocessor:  1024
  Maximum number of threads per block:           512
  Maximum sizes of each dimension of a block:    512 x 512 x 64
  Maximum sizes of each dimension of a grid:     65535 x 65535 x 1
  Maximum memory pitch:                          2147483647 bytes
  Texture alignment:                             256 bytes
  Concurrent copy and kernel execution:          Yes with 1 copy engine(s)
  Run time limit on kernels:                     Yes
  Integrated GPU sharing Host Memory:            No
  Support host page-locked memory mapping:       Yes
  Alignment requirement for Surfaces:            Yes
  Device has ECC support:                        Disabled
  CUDA Device Driver Mode (TCC or WDDM):         WDDM (Windows Display Driver Model)
  Device supports Unified Addressing (UVA):      No
  Device PCI Bus ID / PCI location ID:           2 / 0
  Compute Mode:
     < Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >

deviceQuery, CUDA Driver = CUDART, CUDA Driver Version = 5.0, CUDA Runtime Version = 5.0, NumDevs = 1, Device0 = GeForce GT 320M
*/


#include "hip/hip_runtime.h"
#include ""
//#include "helper_functions.h"
//#include "hip/hip_runtime_api.h"
#include <stdio.h>


__global__ void freqencyStep1(char *d_dat,int len, int *d_freq)
{//����һ���Ƚ����ݼӺ͵�share memory�У�Ȼ�����ۼӵ��Դ��ϡ�
///����Ҳ�����ַ��������Ƿ���һ��share memory�����á���һ�ַ�������share memory�����ã��ڽ��п����ۼ�ʱ��ֻ��ǰ26���߳����0��127���ۼӡ���
///���������ۼ�ʱ������ۼӵ��Խ����ϣ�Ȼ����д��ʱ�����Ա���bank conflict��
    __shared__ int sfreq[3456];//27*128////share memory���ŷţ�ÿ�߳�27��int.

    for(int i=threadIdx.x ;i< 3456;i += blockDim.x)
        sfreq[i] = 0;////����ա�
    __syncthreads();
    int *myfreq = &sfreq[27*threadIdx.x];
    int gridsize = blockDim.x * gridDim.x;
    for(int i=threadIdx.x + blockIdx.x*blockDim.x; i< len; i += gridsize)
        //if((d_dat[i]>='a')&&(d_dat[i]<='z'))//���ȷ������ֻ��a--z�����԰�ifȥ����
            myfreq[d_dat[i]-'a']++;
    __syncthreads();///���߳�ͳ�Ƶ��Լ���sharememory�С�
    if(threadIdx.x<64)
    {
        for(int i=0;i<26;i++)
           myfreq[i] += sfreq[27*(threadIdx.x+64)+i];
    }
    __syncthreads();
    if(threadIdx.x<32)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x+32)+i];
    }
    __syncthreads();
    if(threadIdx.x<16)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x+16)+i];
    }
    if(threadIdx.x< 8)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x+ 8)+i];
    }
    if(threadIdx.x< 4)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x+ 4)+i];
    }    
    if(threadIdx.x< 2)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x+ 2)+i];
    }
    if(threadIdx.x == 0)
    {
        for(int i=0;i<26;i++)
            myfreq[i] += sfreq[27*(threadIdx.x   )+i];
    }
    __syncthreads();

//    myfreq = &d_freq[blockIdx.x * 26 + blockIdx.x];///��һ���Ľ���ȱ��浽�Դ��С�ÿ��block��0��25���̱߳�������
//    if(threadIdx.x<26)
//        myfreq[i] = sfreq[i];
    if(threadIdx.x<26)///����Կ�֧��ԭ�Ӽӣ�����ʹ��ԭ�Ӽӣ�ֱ�Ӽӵ��Դ��ϡ�������û�еڶ����� 1.1������֧��ȫ���Դ��32λԭ�Ӳ�����
	    atomicAdd(&d_freq[threadIdx.x],sfreq[threadIdx.x]);

}
#if 0
__global__ void frequencyStep2(int *d_freq,int *d_swap,int blocksInStep1)//���ǵ�һ��ʹ�õ�block�ĸ�����������٣���ôֱ����26���̼߳��꼴�ɡ�
{////Ӳ����֧��ԭ�Ӽ�ʱ����һ��Ҫ��д���Դ棬Ȼ��ڶ��������ۼӡ��ڶ���ÿ��blockֻʹ��208���̣߳�8��26�̡߳�
    __shared__ int sfreq[256];

    for(int i=threadIdx.x;i<256;i+=blockDim.x) sfreq[i ] = 0;
    __syncthreads();
	int allNumber = blocksInStep1*26;
	int gridSize = blockDim.x*gridDim.x;
    for(int i=threadIdx.x;i<allNumber;i+=gridSize)
	    sfreq[threadIdx.x ] += d_freq[i];
	__syncthreads();
	if(threadIdx.x<104)
	    sfreq[threadIdx.x]+= sfreq[threadIdx.x+104];
	__syncthreads();
	if(threadIdx.x<52)
	    sfreq[threadIdx.x]+= sfreq[threadIdx.x+52];
	__syncthreads();
	if(threadIdx.x<26)
	{
	    sfreq[threadIdx.x]+= sfreq[threadIdx.x+26];
		////��д���Դ��С�
		d_swap[threadIdx.x+blockIdx.x*26] = sfreq[threadIdx.x];
	}
}
////step2��Ҫ���ü��Σ�ֱ�����8��26��Ƶ�����ڡ�
#endif
void hostCalc(char *dat,int len,int *freqency)
{
    int freque[32];
	memset(freque,0,32*sizeof(int));
	for(int i=0;i<len;i++)
	{
	    if((dat[i]>='a')&&(dat[i]<='z'))
		    freque[dat[i]-'a']++;
	}
	memcpy(freqency,freque,26*sizeof(int));
}

void makeData(char *filename,int len)
{
	if(len<0) {
		fprintf(stdout,"len = %d\n",len);
		return;
	}
	FILE *fp = fopen(filename,"w");
	int len1  = (len-(len&1023)+1024;
	char *dat = new char [len1];
	memset(dat,0,len1);
	srand(0);
	for(int i=0;i<len;i++)
	{
		int x = rand();
		x%=26;
		dat[i] = 'a'+x;
	}
	fwrite(dat,1,len,fp);
	fclose(fp);
}
int main(int argc,char **argv)
{
	if(argc<2)
	{
		fprintf(stdout,"usage: a.out datfile\n");
		return -1;
	}
	FILE *fr = NULL;
	if((fr = fopen(argv[1],"r"))==NULL)
	{
		fprintf(stderr,"can't open file %s\n",argv[1]);
		return -1;
	}
	fseek(fr,0,2);
	int len = ftell(fr);
	rewind(fr);
	len = (len-(len&4095))+4096;
	char *dat = new char[len];
	memset(dat,0,len);
	len = fread(dat,1,len,fr);
	fclose(fr);
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		free(dat);
        return -1;
    }
	char *d_dat;
	int *d_freq;
	int gpuFreq[32];
	int cpuFreq[32];
	
	cudaStatus = hipMalloc((void **)&d_dat,len*sizeof(char));
	if(cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
		free(dat);
        return -1;
    }
	cudaStatus = hipMalloc((void **)&d_freq,32*sizeof(int));
	if(cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
		hipFree(d_dat);
		free(dat);
        return -1;
    }
	hipMemcpy(d_dat,dat,len*sizeof(char),hipMemcpyHostToDevice);
	hipMemset(d_freq,0,32*sizeof(int));
	
	freqencyStep1<<<256,128>>>(d_dat,len,d_freq);
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
		hipFree(d_freq);
		hipFree(d_dat);
		free(dat);
        return -1;
    }
	hipMemcpy(gpuFreq,d_freq,32*sizeof(int),hipMemcpyDeviceToHost);

	hostCalc(dat, len,cpuFreq);
	hipFree(d_freq);
	hipFree(d_dat);
	///check
	if(memcmp(gpuFreq,cpuFreq,26*sizeof(int))!=0)
		fprintf(stdout,"CHECK ERROR\n");
	else
		fprintf(stdout,"CHECK OK\n");

	free(dat);

    return 0;
}
